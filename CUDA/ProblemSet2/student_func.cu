#include "hip/hip_runtime.h"

#include "reference_calc.cpp"
#include "utils.h"

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
    assert(filterWidth % 2 == 1);
  
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ( r >= numRows || c >= numCols ) { 
        return;
    }
    
    float result = 0.f;
      for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
        for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
        
          int image_r = min(max(r + filter_r, 0), static_cast<int>(numRows - 1));
          int image_c = min(max(c + filter_c, 0), static_cast<int>(numCols - 1));

          float image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
          float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];

          result += image_value * filter_value;
        }
      }

      outputChannel[r * numCols + c] = result;
  
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return; 
    
  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
  
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * numRowsImage * numColsImage));
  
  checkCudaErrors(hipMemcpy(d_filter,h_filter, sizeof(float)*numRowsImage*numColsImage, hipMemcpyHostToDevice));

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  const dim3 blockSize(32,32,1);
  const dim3 gridSize(numCols/16, numRows/16, 1);


  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,numRows,numCols,d_red,d_green,d_blue);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  gaussian_blur<<<gridSize, blockSize>>>(d_red,d_redBlurred,numRows,numCols,d_filter,filterWidth);
  gaussian_blur<<<gridSize, blockSize>>>(d_green,d_greenBlurred,numRows,numCols,d_filter,filterWidth);
  gaussian_blur<<<gridSize, blockSize>>>(d_blue,d_blueBlurred,numRows,numCols,d_filter,filterWidth);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,d_greenBlurred,d_blueBlurred,d_outputImageRGBA,numRows,numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}
