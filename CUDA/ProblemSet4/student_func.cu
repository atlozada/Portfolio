#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void getHistogram(   unsigned int * d_bins,
                                unsigned int* const d_inputVals,
                                unsigned int iteration,
                                const int numElems) {

    __shared__ unsigned int* bins;
    __shared__ unsigned int* inputVals;

    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx >= numElems) {
        return;
    }

    bins = d_bins;
    inputVals = d_inputVals;
    int bin;

    if ((inputVals[idx] & (1<<iteration)) == (1<<iteration)) {
        bin = 1;
    } else {
        bin = 0;
    }

    if (bin == 1) {
         atomicAdd(&bins[1], 1);
    } else {
         atomicAdd(&bins[0], 1);
    }

    if (idx == 1) {
        d_bins = bins;
    }

}

__global__ void prefixScan( unsigned int * d_inputVals,
                            unsigned int * d_binScan,
                            unsigned int blockSize,
                            const int numElems,
                            unsigned int outerIteration,
                            unsigned int innerIteration) {

    __shared__ unsigned int* inputVals;
    __shared__ unsigned int* binScan;

    int idx = threadIdx.x + blockSize * innerIteration;

    if(idx >= numElems) {
        return;
    }

    unsigned int sum = 0;
    inputVals = d_inputVals;
    binScan = d_binScan;

    if (idx > 0 && (inputVals[idx-1] & (1<<outerIteration))  == (1<<outerIteration)) {
        sum = 1;
    }

    binScan[idx] = sum;

    __syncthreads();

    for(int n = 1; n <= blockSize; n *= 2) {
        int prefix = idx - n;

        if (prefix >= 0 && prefix >= blockSize*innerIteration) {
            sum = binScan[prefix];
        }

        __syncthreads();

        if (prefix >= 0 && prefix >= blockSize*innerIteration) {
            binScan[idx] += sum;
        }
        __syncthreads();
    }

    if(innerIteration > 0) {
        binScan[idx] += binScan[innerIteration*blockSize - 1];
    }

    __syncthreads();

    if (idx == 1) {
        d_binScan = binScan;
    }
}

__global__ void calcOffset( unsigned int* const d_inputVals,
                            unsigned int* const d_inputPos,
                            unsigned int* const d_binScan,
                            unsigned int* d_outputVals,
                            unsigned int* d_outputPos,
                            unsigned int* d_offsetPos,
                            unsigned int numOnes,
                            const size_t numElems,
                            unsigned int iteration) {

    __shared__ unsigned int* inputVals;
    __shared__ unsigned int* inputPos;
    __shared__ unsigned int* outputVals;
    __shared__ unsigned int* outputPos;
    __shared__ unsigned int* offsetPos;
    __shared__ unsigned int* binScan;

    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx >= numElems) {
        return;
    }

    inputVals = d_inputVals;
    inputPos = d_inputPos;
    outputVals = d_outputVals;
    outputPos = d_outputPos;
    offsetPos = d_offsetPos;
    binScan = d_binScan;

    unsigned int offsetSum;
    unsigned int baseOffset;

    if( ( inputVals[idx] & (1<<iteration)) == (1<<iteration)) {
        offsetSum = binScan[idx];
        baseOffset = numOnes;
    } else {
        offsetSum = (idx) - binScan[idx];
        baseOffset = 0;
    }

    offsetPos[idx] = baseOffset+offsetSum;
    outputPos[baseOffset+offsetSum]  = inputPos[idx];
    outputVals[baseOffset+offsetSum] = inputVals[idx];

    if (idx == 1) {
        d_offsetPos = offsetPos;
        d_outputVals = outputVals;
        d_outputPos = outputPos;
    }
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems) {

    unsigned int* d_histogramBins;
    unsigned int  h_histogramBins[2];
    unsigned int* d_binScan;
    unsigned int* d_offsetPos;

    checkCudaErrors(hipMalloc(&d_histogramBins, sizeof(int)*2));
    checkCudaErrors(hipMalloc(&d_binScan, sizeof(int)*numElems));
    checkCudaErrors(hipMalloc(&d_offsetPos, sizeof(int)*numElems));


    dim3 blocksize(1024);
    dim3 gridsize(numElems/1024+1);


    for(unsigned int i = 0; i < 32; i++) {

        checkCudaErrors(hipMemset(d_histogramBins, 0, sizeof(int)*2));
        checkCudaErrors(hipMemset(d_binScan, 0, sizeof(int)*numElems));
        checkCudaErrors(hipMemset(d_outputVals, 0, sizeof(int)*numElems));
        checkCudaErrors(hipMemset(d_outputPos, 0, sizeof(int)*numElems));

        getHistogram<<<blocksize, gridsize>>>(d_histogramBins, d_inputVals, i, numElems);

        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipMemcpy(&h_histogramBins, d_histogramBins, sizeof(int)*2, hipMemcpyDeviceToHost));


        for(int j = 0; j < gridsize.x; j++) {
            prefixScan<<<dim3(1), blocksize>>>(d_inputVals, d_binScan, blocksize.x, numElems, i, j);
            hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        }

        calcOffset<<<gridsize, blocksize>>>(d_inputVals, d_inputPos, d_binScan, d_outputVals, d_outputPos, d_offsetPos, h_histogramBins[0], numElems, i);

        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipMemcpy(d_inputVals, d_outputVals, sizeof(int)*numElems, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(d_inputPos, d_outputPos, sizeof(int)*numElems, hipMemcpyDeviceToDevice));
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    }
}
