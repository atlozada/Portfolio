#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  
  size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  size_t idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  
  uchar4 rgba = rgbaImage[idx * numRows + idy];
  float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
  greyImage[idx * numRows + idy] = channelSum;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  const dim3 blockSize(17, 17, 1);  //TODO
  const dim3 gridSize(numCols/16, numRows/16, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}